
#include <hip/hip_runtime.h>
#include <algorithm>
#include <array>
#include <chrono>
#include <cstddef>
#include <iomanip>
#include <iostream>
#include <iterator>
#include <random>
#include <stdexcept>
#include <string>
#include <thread>
#include <utility>
#include <vector>

__global__ void translatory_motion(double *arr_position, double *new_position, double *arr_speed,
                                   int size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < size; i += stride) {

    new_position[i] += arr_position[i] + arr_speed[i];
  }
}

class RunTimeCalculator {
public:
  void start()
  {
    start_time = std::chrono::high_resolution_clock::now();
  }
  double getTime()
  {
    auto elapsed_time = std::chrono::duration_cast<std::chrono::duration<double, std::ratio<1>>>(
                          std::chrono::high_resolution_clock::now() - start_time)
                          .count();
    return elapsed_time;
  }

private:
  std::chrono::time_point<std::chrono::high_resolution_clock> start_time;
};

class RandomNumberGenerator {
public:
  RandomNumberGenerator()
  {
  }

  static inline double getRandom(double begin_range, double end_range)
  {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(begin_range, end_range);

    return dis(gen);
  }
};

class Sport {
private:
  unsigned int distance;

public:
  Sport(unsigned int _distance) : distance{ _distance }
  {
  }
  unsigned int getDistance() const
  {
    return distance;
  }
};

class Swimming : public Sport {
public:
  Swimming(unsigned int _distance) : Sport{ _distance }
  {
  }
};

class Cycling : public Sport {
public:
  Cycling(unsigned int _distance) : Sport{ _distance }
  {
  }
};

class Run : public Sport {
public:
  Run(unsigned int _distance) : Sport{ _distance }
  {
  }
};

class Triathlon {
private:
  std::vector<Sport> sport_set;

public:
  Triathlon(unsigned int swim_lane_length, unsigned int velodrom_length, unsigned int runway_length)
  {
    sport_set.push_back(Swimming(swim_lane_length));
    sport_set.push_back(Cycling(velodrom_length));
    sport_set.push_back(Run(runway_length));
  }

  void print_distance()
  {
    for (const auto &sport : sport_set) {
      std::cout << sport.getDistance() << "\n";
    }
  }

  const std::vector<Sport> getStages()
  {
    return sport_set;
  }
};

struct CompleteRace {
  std::string name;
  int team_name;
  double total_time;
};

class Sporter {
private:
  std::string name;

public:
  Sporter(std::string _name) : name(_name)
  {
  }
  const std::string getName() const
  {
    return name;
  }
};

class Competitor : public Sporter {
private:
  bool complete;
  int stage;
  int standby_time;
  int team_name;
  double prev_position;
  double position;
  double speed;
  double total_time;

public:
  Competitor(int _team_name, std::string _name, double _speed)
      : Sporter(_name), speed(_speed), team_name(_team_name)
  {
    position = 0.0;
    prev_position = 0.0;
    stage = 0;
    complete = false;
    standby_time = 0;
    total_time = 0.0;
  }

  double getPosition() const
  {
    return position;
  }
  int getTeamName() const
  {
    return team_name;
  }
  void setPosition(double new_position)
  {
    prev_position = position;
    position = new_position;
  }

  double getPrevPosition() const
  {
    return prev_position;
  }

  double getSpeed() const
  {
    return speed;
  }

  void setSpeed(double new_speed)
  {
    speed = new_speed;
  }

  int getStage() const
  {
    return stage;
  }

  void increaseStage()
  {
    stage++;
  }

  bool getComplete() const
  {
    return complete;
  }

  void setComplete()
  {
    complete = true;
  }
  void decreaseStandbyTime()
  {
    standby_time--;
  }
  int getStandbyTime() const
  {
    return standby_time;
  }
  void setStandbyTime(int a)
  {
    standby_time = a;
  }
  double getTotalTime() const
  {
    return total_time;
  }
  void increaseTotalTime()
  {
    total_time += 1.0;
  }
  void setTotalTime(double new_total_time)
  {
    total_time = new_total_time;
  }
};

class TriathlonTeam {
private:
  std::array<Competitor, 3> team;

public:
  TriathlonTeam(Competitor c1, Competitor c2, Competitor c3) : team{ c1, c2, c3 } {};

  std::array<Competitor, 3> getTeam() const
  {
    return team;
  }

  bool setTeamNewStats(std::vector<double> vec, int team_name, Triathlon &t,
                       std::vector<CompleteRace> &sorted)
  {
    bool first_print = false;
    for (size_t i = 0; i < 3; i++) {
      if (!team[i].getComplete()) {
        if ((team[i].getStage() == 0 && vec[i] >= t.getStages()[0].getDistance())) {
          team[i].increaseStage();
          team[i].setStandbyTime(10);
          team[i].setSpeed(team[i].getSpeed() * 3);
        } else if (team[i].getStage() == 1
                   && vec[i] >= t.getStages()[0].getDistance() + t.getStages()[1].getDistance()) {
          team[i].increaseStage();
          team[i].setStandbyTime(10);
          team[i].setSpeed(team[i].getSpeed() / 9);
        } else if (team[i].getStage() == 2
                   && vec[i] >= t.getStages()[0].getDistance() + t.getStages()[1].getDistance()
                                  + t.getStages()[2].getDistance()) {
          team[i].increaseStage();
          team[i].setComplete();
          if (sorted.size() == 0) {
            first_print = true;
          }
          sorted.push_back({ team[i].getName(), team_name, team[i].getTotalTime() });
        }
        if (team[i].getStandbyTime() != 0) {
          team[i].decreaseStandbyTime();
        } else {
          team[i].setPosition(vec[i]);
        }
        team[i].increaseTotalTime();
      }
    }
    return first_print;
  }
};

bool compareTeamName(const CompleteRace &first, const CompleteRace &second)
{
  return first.team_name < second.team_name;
}
bool compareTotalTime(const CompleteRace &first, const CompleteRace &second)
{
  return first.total_time < second.total_time;
}

void runTimeGetCompetitor(std::vector<TriathlonTeam> &teams)
{
  while (true) {
    std::string input;
    std::cin >> input;
    std::vector<std::string> seperated_input;

    auto split_string = [&seperated_input](const std::string &str) {
      std::istringstream iss(str);
      std::string token;

      while (iss >> std::quoted(token)) {
        seperated_input.push_back(token);
      }
    };

    split_string(input);
    std::cout << "Requested Inputs:\n";
    for (const auto &team : teams) {
      for (const auto &competitor : team.getTeam()) {
        for (const auto &i : seperated_input) {
          if (i == competitor.getName()) {
            std::cout << " Team Name: " << competitor.getTeamName()
                      << " Name: " << competitor.getName()
                      << " Position: " << competitor.getPosition()
                      << " Speed: " << competitor.getSpeed() << "\n";
          }
        }
      }
    }
  }
}

class Race {
private:
  std::vector<TriathlonTeam> teams;
  Triathlon stages;
  std::vector<CompleteRace> sorted;

public:
  Race(unsigned int stage1, unsigned int stage2, unsigned int stage3)
      : stages{ stage1, stage2, stage3 }
  {
  }
  void InitRace(const std::vector<std::string> sporter_list, double speed_range_begin,
                double speed_range_end)
  {
    if (sporter_list.size()) {
      if (sporter_list.size() % 3 != 0) {
        throw std::runtime_error("Missing team member.");
      }
      teams.reserve(sporter_list.size() / 3);
    } else {
      throw std::runtime_error("No team members.");
    }
    int team_count = 0;
    for (size_t i = 0; i < sporter_list.size(); i += 3) {

      teams.emplace_back(TriathlonTeam{
        Competitor{ team_count, sporter_list[i],
                    RandomNumberGenerator::getRandom(speed_range_begin, speed_range_end) },
        Competitor{ team_count, sporter_list[i + 1],
                    RandomNumberGenerator::getRandom(speed_range_begin, speed_range_end) },
        Competitor{ team_count, sporter_list[i + 2],
                    RandomNumberGenerator::getRandom(speed_range_begin, speed_range_end) } });
      team_count++;
    }
  }

  void RaceStart()
  {
    std::thread input_t([this]() { runTimeGetCompetitor(teams); });

    double *d_position;
    double *d_new_position;
    double *d_speed;

    size_t COMPETITOR_SIZE = teams.size() * teams[0].getTeam().size();

    hipMalloc((void **)&d_position, COMPETITOR_SIZE * sizeof(double));
    hipMalloc((void **)&d_new_position, COMPETITOR_SIZE * sizeof(double));
    hipMalloc((void **)&d_speed, COMPETITOR_SIZE * sizeof(double));
    while (true) {

      if (sorted.size() == teams.size() * teams[0].getTeam().size()) {
        input_t.detach();
        break;
      }
      std::vector<double> temp_competitor_position;
      std::vector<double> temp_competitor_new_position(COMPETITOR_SIZE, 0.0);
      std::vector<double> temp_competitor_speed;

      for (const auto &team : teams) {
        for (const auto &competitor : team.getTeam()) {
          temp_competitor_position.push_back(competitor.getPosition());
          temp_competitor_speed.push_back(competitor.getSpeed());
        }
      }

      hipMemcpy(d_position, temp_competitor_position.data(), COMPETITOR_SIZE * sizeof(double),
                 hipMemcpyHostToDevice);

      hipMemcpy(d_new_position, temp_competitor_new_position.data(),
                 COMPETITOR_SIZE * sizeof(double), hipMemcpyHostToDevice);

      hipMemcpy(d_speed, temp_competitor_speed.data(), COMPETITOR_SIZE * sizeof(double),
                 hipMemcpyHostToDevice);

      int block_size = 256;
      int num_blocks = (COMPETITOR_SIZE + block_size - 1) / block_size;

      translatory_motion<<<num_blocks, block_size>>>(d_position, d_new_position, d_speed,
                                                     COMPETITOR_SIZE);
      hipDeviceSynchronize();

      hipMemcpy(temp_competitor_position.data(), d_position, COMPETITOR_SIZE * sizeof(double),
                 hipMemcpyDeviceToHost);
      hipMemcpy(temp_competitor_new_position.data(), d_new_position,
                 COMPETITOR_SIZE * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(temp_competitor_speed.data(), d_speed, COMPETITOR_SIZE * sizeof(double),
                 hipMemcpyDeviceToHost);

      divideOfThree(temp_competitor_new_position);
      std::this_thread::sleep_for(std::chrono::seconds(1));
    }
    hipFree(d_position);
    hipFree(d_new_position);
    hipFree(d_speed);

    printResult();
  }

  void printStats()
  {
    std::cout << std::setw(13) << std::left << "Name" << std::setw(13) << std::left << "Speed"
              << std::setw(13) << std::left << "Position" << std::setw(13) << std::left
              << "PrevPosition" << std::setw(13) << std::left << "Stage" << std::setw(13)
              << std::left << "Standby Time" << std::setw(13) << std::left << "Complete"
              << std::setw(13) << std::left << "Total Time"
              << "\n";
    for (const auto &team : teams) {
      for (const auto &s : team.getTeam()) {
        std::cout << std::setw(13) << std::left << s.getName() << std::setw(13) << std::left
                  << s.getSpeed() << std::setw(13) << std::left << s.getPosition() << std::setw(13)
                  << std::left << s.getPrevPosition() << std::setw(13) << std::left << s.getStage()
                  << std::setw(13) << std::left << s.getStandbyTime() << std::setw(13) << std::left
                  << s.getComplete() << std::setw(13) << std::left << s.getTotalTime() << "\n";
      }
    }
  }
  void printResult()
  {
    std::cout << "\n"
              << "-------------- RESULT --------------\n";
    std::sort(sorted.begin(), sorted.end(), compareTotalTime);
    std::cout << "BEST SPORTER => " << sorted[0].name << " | Team Name: " << sorted[0].team_name
              << " | Total Time: " << sorted[0].total_time << "\n";
    std::sort(sorted.begin(), sorted.end(), compareTeamName);
    std::pair<int, double> best{ std::make_pair<int, double>(-1, 0.0) };

    for (size_t i = 0; i < sorted.size(); i += 3) {
      double sum = sorted[i].total_time + sorted[i + 1].total_time + sorted[i + 2].total_time;
      if (best.second == 0.0) {
        best.first = 0;
        best.second = sum;
      }
      if (sum < best.second) {
        best.first = sorted[i].team_name, best.second = sum;
      }
    }
    std::cout << "BEST TEAM  => "
              << "Team Name: " << best.first << " | Total Time: " << best.second << "\n";
  }

private:
  void divideOfThree(std::vector<double> &temp_new_position)
  {
    auto it = temp_new_position.begin();
    size_t team_count = 0;
    while (it != temp_new_position.end()) {
      std::vector<double> group;
      group.reserve(3);

      for (int i = 0; i < 3 && it != temp_new_position.end(); ++i, ++it) {
        group.push_back(*it);
      }

      if (teams[team_count].setTeamNewStats(group, team_count, stages, sorted)) {
        std::cout << "Firt to Finish =>  Name: " << sorted[0].name
                  << " | Team Name: " << sorted[0].team_name
                  << " | Total Time: " << sorted[0].total_time << "\n\n";
      }
      team_count++;
      if (sorted.size() == teams.size() * teams[0].getTeam().size()) {
        printStats();
      }
    }
  }
};

int main()
{
  std::vector<std::string> sporter_name{
    "s1",   "s2",   "s3",   "s4",   "s5",   "s6",   "s7",   "s8",   "s9",   "s10",  "s11",  "s12",
    "s13",  "s14",  "s15",  "s16",  "s17",  "s18",  "s19",  "s20",  "s21",  "s22",  "s23",  "s24",
    "s25",  "s26",  "s27",  "s28",  "s29",  "s30",  "s31",  "s32",  "s33",  "s34",  "s35",  "s36",
    "s37",  "s38",  "s39",  "s40",  "s41",  "s42",  "s43",  "s44",  "s45",  "s46",  "s47",  "s48",
    "s49",  "s50",  "s51",  "s52",  "s53",  "s54",  "s55",  "s56",  "s57",  "s58",  "s59",  "s60",
    "s61",  "s62",  "s63",  "s64",  "s65",  "s66",  "s67",  "s68",  "s69",  "s70",  "s71",  "s72",
    "s73",  "s74",  "s75",  "s76",  "s77",  "s78",  "s79",  "s80",  "s81",  "s82",  "s83",  "s84",
    "s85",  "s86",  "s87",  "s88",  "s89",  "s90",  "s91",  "s92",  "s93",  "s94",  "s95",  "s96",
    "s97",  "s98",  "s99",  "s100", "s101", "s102", "s103", "s104", "s105", "s106", "s107", "s108",
    "s109", "s110", "s111", "s112", "s113", "s114", "s115", "s116", "s117", "s118", "s119", "s120",
    "s121", "s122", "s123", "s124", "s125", "s126", "s127", "s128", "s129", "s130", "s131", "s132",
    "s133", "s134", "s135", "s136", "s137", "s138", "s139", "s140", "s141", "s142", "s143", "s144",
    "s145", "s146", "s147", "s148", "s149", "s150", "s151", "s152", "s153", "s154", "s155", "s156",
    "s157", "s158", "s159", "s160", "s161", "s162", "s163", "s164", "s165", "s166", "s167", "s168",
    "s169", "s170", "s171", "s172", "s173", "s174", "s175", "s176", "s177", "s178", "s179", "s180",
    "s181", "s182", "s183", "s184", "s185", "s186", "s187", "s188", "s189", "s190", "s191", "s192",
    "s193", "s194", "s195", "s196", "s197", "s198", "s199", "s200", "s201", "s202", "s203", "s204",
    "s205", "s206", "s207", "s208", "s209", "s210", "s211", "s212", "s213", "s214", "s215", "s216",
    "s217", "s218", "s219", "s220", "s221", "s222", "s223", "s224", "s225", "s226", "s227", "s228",
    "s229", "s230", "s231", "s232", "s233", "s234", "s235", "s236", "s237", "s238", "s239", "s240",
    "s241", "s242", "s243", "s244", "s245", "s246", "s247", "s248", "s249", "s250", "s251", "s252",
    "s253", "s254", "s255", "s256", "s257", "s258", "s259", "s260", "s261", "s262", "s263", "s264",
    "s265", "s266", "s267", "s268", "s269", "s270", "s271", "s272", "s273", "s274", "s275", "s276",
    "s277", "s278", "s279", "s280", "s281", "s282", "s283", "s284", "s285", "s286", "s287", "s288",
    "s289", "s290", "s291", "s292", "s293", "s294", "s295", "s296", "s297", "s298", "s299", "s300",
    "s301", "s302", "s303", "s304", "s305", "s306", "s307", "s308", "s309", "s310", "s311", "s312",
    "s313", "s314", "s315", "s316", "s317", "s318", "s319", "s320", "s321", "s322", "s323", "s324",
    "s325", "s326", "s327", "s328", "s329", "s330", "s331", "s332", "s333", "s334", "s335", "s336",
    "s337", "s338", "s339", "s340", "s341", "s342", "s343", "s344", "s345", "s346", "s347", "s348",
    "s349", "s350", "s351", "s352", "s353", "s354", "s355", "s356", "s357", "s358", "s359", "s360",
    "s361", "s362", "s363", "s364", "s365", "s366", "s367", "s368", "s369", "s370", "s371", "s372",
    "s373", "s374", "s375", "s376", "s377", "s378", "s379", "s380", "s381", "s382", "s383", "s384",
    "s385", "s386", "s387", "s388", "s389", "s390", "s391", "s392", "s393", "s394", "s395", "s396",
    "s397", "s398", "s399", "s400", "s401", "s402", "s403", "s404", "s405", "s406", "s407", "s408",
    "s409", "s410", "s411", "s412", "s413", "s414", "s415", "s416", "s417", "s418", "s419", "s420",
    "s421", "s422", "s423", "s424", "s425", "s426", "s427", "s428", "s429", "s430", "s431", "s432",
    "s433", "s434", "s435", "s436", "s437", "s438", "s439", "s440", "s441", "s442", "s443", "s444",
    "s445", "s446", "s447", "s448", "s449", "s450", "s451", "s452", "s453", "s454", "s455", "s456",
    "s457", "s458", "s459", "s460", "s461", "s462", "s463", "s464", "s465", "s466", "s467", "s468",
    "s469", "s470", "s471", "s472", "s473", "s474", "s475", "s476", "s477", "s478", "s479", "s480",
    "s481", "s482", "s483", "s484", "s485", "s486", "s487", "s488", "s489", "s490", "s491", "s492",
    "s493", "s494", "s495", "s496", "s497", "s498", "s499", "s500", "s501", "s502", "s503", "s504",
    "s505", "s506", "s507", "s508", "s509", "s510", "s511", "s512", "s513", "s514", "s515", "s516",
    "s517", "s518", "s519", "s520", "s521", "s522", "s523", "s524", "s525", "s526", "s527", "s528",
    "s529", "s530", "s531", "s532", "s533", "s534", "s535", "s536", "s537", "s538", "s539", "s540",
    "s541", "s542", "s543", "s544", "s545", "s546", "s547", "s548", "s549", "s550", "s551", "s552",
    "s553", "s554", "s555", "s556", "s557", "s558", "s559", "s560", "s561", "s562", "s563", "s564",
    "s565", "s566", "s567", "s568", "s569", "s570", "s571", "s572", "s573", "s574", "s575", "s576",
    "s577", "s578", "s579", "s580", "s581", "s582", "s583", "s584", "s585", "s586", "s587", "s588",
    "s589", "s590", "s591", "s592", "s593", "s594", "s595", "s596", "s597", "s598", "s599", "s600",
    "s601", "s602", "s603", "s604", "s605", "s606", "s607", "s608", "s609", "s610", "s611", "s612",
    "s613", "s614", "s615", "s616", "s617", "s618", "s619", "s620", "s621", "s622", "s623", "s624",
    "s625", "s626", "s627", "s628", "s629", "s630", "s631", "s632", "s633", "s634", "s635", "s636",
    "s637", "s638", "s639", "s640", "s641", "s642", "s643", "s644", "s645", "s646", "s647", "s648",
    "s649", "s650", "s651", "s652", "s653", "s654", "s655", "s656", "s657", "s658", "s659", "s660",
    "s661", "s662", "s663", "s664", "s665", "s666", "s667", "s668", "s669", "s670", "s671", "s672",
    "s673", "s674", "s675", "s676", "s677", "s678", "s679", "s680", "s681", "s682", "s683", "s684",
    "s685", "s686", "s687", "s688", "s689", "s690", "s691", "s692", "s693", "s694", "s695", "s696",
    "s697", "s698", "s699", "s700", "s701", "s702", "s703", "s704", "s705", "s706", "s707", "s708",
    "s709", "s710", "s711", "s712", "s713", "s714", "s715", "s716", "s717", "s718", "s719", "s720",
    "s721", "s722", "s723", "s724", "s725", "s726", "s727", "s728", "s729", "s730", "s731", "s732",
    "s733", "s734", "s735", "s736", "s737", "s738", "s739", "s740", "s741", "s742", "s743", "s744",
    "s745", "s746", "s747", "s748", "s749", "s750", "s751", "s752", "s753", "s754", "s755", "s756",
    "s757", "s758", "s759", "s760", "s761", "s762", "s763", "s764", "s765", "s766", "s767", "s768",
    "s769", "s770", "s771", "s772", "s773", "s774", "s775", "s776", "s777", "s778", "s779", "s780",
    "s781", "s782", "s783", "s784", "s785", "s786", "s787", "s788", "s789", "s790", "s791", "s792",
    "s793", "s794", "s795", "s796", "s797", "s798", "s799", "s800", "s801", "s802", "s803", "s804",
    "s805", "s806", "s807", "s808", "s809", "s810", "s811", "s812", "s813", "s814", "s815", "s816",
    "s817", "s818", "s819", "s820", "s821", "s822", "s823", "s824", "s825", "s826", "s827", "s828",
    "s829", "s830", "s831", "s832", "s833", "s834", "s835", "s836", "s837", "s838", "s839", "s840",
    "s841", "s842", "s843", "s844", "s845", "s846", "s847", "s848", "s849", "s850", "s851", "s852",
    "s853", "s854", "s855", "s856", "s857", "s858", "s859", "s860", "s861", "s862", "s863", "s864",
    "s865", "s866", "s867", "s868", "s869", "s870", "s871", "s872", "s873", "s874", "s875", "s876",
    "s877", "s878", "s879", "s880", "s881", "s882", "s883", "s884", "s885", "s886", "s887", "s888",
    "s889", "s890", "s891", "s892", "s893", "s894", "s895", "s896", "s897", "s898", "s899", "s900"
  };

  Race r{ 10, 20, 6 };
  std::cout << "--------------------------- INIT RACE ---------------------------\n";
  r.InitRace(sporter_name, 1.0, 5.0);
  std::cout << "--------------------------- START RACE --------------------------\n";
  r.RaceStart();
  return 0;
}
